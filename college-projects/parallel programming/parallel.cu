#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define V 4
#define MAXWT 100
#define inf 999

__global__ void closestNode(int *node_dist,int *visited_node,int* global_closest)
{
	int dist = inf+1;
	int node = -1;
    int i;

    for (i = 0; i < V; i++) {
        if ((node_dist[i] < dist) && (visited_node[i] != 1)) {
            dist = node_dist[i];
            node = i;
        }
    }

    global_closest[0] = node;
    visited_node[node] = 1;
}
__global__ void relax(int *graph,int *node_dist,int* parent_node,int* visited_node,int* global_closest)
{
    int next = threadIdx.x;
    int source = global_closest[0];

    int edge = graph[source*V + next];
    int new_dist = node_dist[source] + edge;

    if ((edge != 0) &&
        (visited_node[next] != 1) &&
        (new_dist < node_dist[next])) {
        node_dist[next] = new_dist;
        parent_node[next] = source;
    }

}

void bellman_ford(int *graph,int k,int edge[][2],int *node_dist,int *parent_node)
{
    int i,u,v,j,origin=0,flag=1;
    //printf("origin = %d\n",origin);
    node_dist[origin]=0;
    for(i=0;i<V-1;i++)
    {
        for(j=0;j<k;j++)
        {
            u = edge[j][0];
            v = edge[j][1];
            if(node_dist[u]+graph[u*V+v] < node_dist[v])
            {   node_dist[v] = node_dist[u] + graph[u*V+v];
                parent_node[v]=u ;
            }
        }
    }
    for(j=0;j<k;j++)
    {
            u = edge[j][0];
            v = edge[j][1] ;
            if(node_dist[u]+graph[u*V+v] < node_dist[v])
                flag = 0 ;
    }
   /* if(flag)
        for(i=0;i<V;i++)
            printf("Vertex %d cost = %d parent = %d\n",i,node_dist[i],parent_node[i]);*/

}

int main()
{
	int graph_size = V*V*sizeof(int);
	int int_array = V*sizeof(int);
	//int data_array = VERTICES*sizeof(int);
	//int *graph = (int *)malloc(graph_size);
	int *node_dist = (int *)malloc(int_array);
	int *parent_node = (int *)malloc(int_array);
	int *visited_node = (int *)malloc(int_array);

	//int *pnmatrix = (int *)malloc(int_array);
	//int *distmatrix = (int *)malloc(int_array);

	int graph[V*V]={0,-5,2,3,0,0,4,0,0,0,0,1,0,0,0,0};
	/*0 -5 2 3
	  0 0 4 0
	  0 0 0 1
	  0 0 0 0
	 */
	int edge[20][2]={0};
	int i,j,k=0;

	double ts;
	clock_t t1,t2;
	for(i=0;i<V;i++)
    {
        for(j=0;j<V;j++)
        {
            if(graph[i*V+j]!=0)
            {
                edge[k][0]=i;
                edge[k++][1]=j;
            }
        }
    }
	printf("graph adjacency matrix:\n");
    for(i=0;i<V;i++)
    {
        for(j=0;j<V;j++)
        {
            printf("%d ",graph[i*V+j]);
        }
        printf("\n");
    }
    printf("\n");
    for(i=0;i<k;i++)
    {
        printf("%d %d\n",edge[i][0],edge[i][1]);
    }

	printf("Variables created, allocated.\n");

	int *gpu_graph,*gpu_node_dist,*gpu_parent,*gpu_visited;
	hipMalloc((void **)&gpu_graph,graph_size);
	hipMalloc((void **)&gpu_node_dist,int_array);
	hipMalloc((void **)&gpu_parent,int_array);
	hipMalloc((void **)&gpu_visited,int_array);
	int *closest_vertex = (int*)malloc(sizeof(int));
	int* gpu_closest_vertex;
    closest_vertex[0] = -1;
    hipMalloc((void **)&gpu_closest_vertex,sizeof(int));
    hipMemcpy(gpu_closest_vertex,closest_vertex,sizeof(int),hipMemcpyHostToDevice);

    for(i=0;i<V;i++)
    {
    	node_dist[i]=inf;
    	parent_node[i]=-1;
    	visited_node[i]=0;
    }

	printf("Variables initialized.\n");

	int origin = 0;
	printf("origin = %d\n",origin);

	t1=clock();
	bellman_ford(graph,k,edge,node_dist,parent_node);

	printf("modified graph:\n");
	for(i=0;i<V;i++)
    {
    	for(j=0;j<V;j++)
    	{
    		if(graph[i*V+j]!=0)
    		{graph[i*V+j] = graph[i*V+j] + node_dist[i] - node_dist[j];}
    		printf("%d ",graph[i*V+j]);
    	}
    	printf("\n");
    }
	t2=clock();
	ts=((double)t2-t1)/CLOCKS_PER_SEC;
    /*for(i=0;i<V;i++)
	{
		printf("%d %d %d\n",i,node_dist[i],parent_node[i]);
	}*/
    for(i=0;i<V;i++)
    {
    	node_dist[i]=inf;
    	parent_node[i]=-1;
    	visited_node[i]=0;
    }

	hipEvent_t start,stop;
	float t;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	node_dist[origin]=0;

	hipMemcpy(gpu_graph,graph,graph_size,hipMemcpyHostToDevice);
	hipMemcpy(gpu_node_dist,node_dist,int_array,hipMemcpyHostToDevice);
	hipMemcpy(gpu_parent,parent_node,int_array,hipMemcpyHostToDevice);
	hipMemcpy(gpu_visited,visited_node,int_array,hipMemcpyHostToDevice);

	dim3 ngrid(1,1,1);
	dim3 nblock(V,1,1);

	hipEventRecord(start);
	for(i=0;i<V;i++)
	{
		closestNode<<<1,1>>>(gpu_node_dist,gpu_visited,gpu_closest_vertex);
		relax<<<ngrid,nblock>>>(gpu_graph,gpu_node_dist,gpu_parent,gpu_visited,gpu_closest_vertex);
	}
	hipEventRecord(stop);

	hipMemcpy(node_dist, gpu_node_dist, int_array, hipMemcpyDeviceToHost);
	hipMemcpy(parent_node,gpu_parent,int_array,hipMemcpyDeviceToHost);
	hipMemcpy(visited_node,gpu_visited,int_array, hipMemcpyDeviceToHost);
	/*for (i = 0; i < V; i++)
	{
        pnmatrix[i] = parent_node[i];
        distmatrix[i] = node_dist[i];
    }*/
	printf("\n");
	for(i=0;i<V;i++)
	{
		printf("%d %d\n",i,node_dist[i]);
	}

    hipFree(gpu_graph);
    hipFree(gpu_node_dist);
    hipFree(gpu_parent);
    hipFree(gpu_visited);


    printf("vertices = %d\n",V);
    hipEventElapsedTime(&t,start,stop);
    printf("CUDA Time (ms): %f\n",t);
    ts=ts*1000;
    printf("full code time (s): %f\n",ts+t);
	return 0;
}

